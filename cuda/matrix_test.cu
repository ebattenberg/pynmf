#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "matrix.h"

#define EPS 2.2204E-16


__global__ void vecEps(float* a,const int N);
__global__ void vecDiv(float* a,float* b,float* c,const int N);
__global__ void vecMult(float* a,float* b,float* c,const int N);
__global__ void colDiv(float* a, float* b, float* c, int M, int N);
__global__ void colMul(float* a, float* b, float* c, int M, int N);
__global__ void rowDiv(float* a, float* b, float* c, int M, int N);
template <unsigned int blockSize>
__global__ void reduce2D(float *g_idata, float *g_odata, int N);
template <unsigned int blockSize>
__global__ void reduce2DStrided(float *g_idata, float *g_odata, int N, int stride);
template <unsigned int blockSize>
__global__ void reduce1DDiff(float *g_idata1, float *g_idata2, float *g_odata, int N);
template <unsigned int blockSize>
__global__ void reduce1DDiv(float *g_idata1, float *g_idata2, float *g_odata, int N);
template <unsigned int blockSize>
__global__ void reduce1D(float *g_idata, float *g_odata, int N);
template <unsigned int blockSize>
__global__ void reduce1D_rolled(float *g_idata, float *g_odata, int N);
template <unsigned int blockSize>
__global__ void reduce1D_rolled2(float *g_idata, float *g_odata, int N);
template <unsigned int blockSize>
__global__ void reduce1D_strided(float *g_idata, float *g_odata, int N);
template <unsigned int blockSize>
__global__ void reduce1D_interleaved(float *g_idata, float *g_odata, int N);


void read_matrix(matrix* A, char* file){
    //read matrix in from file, store in column-major order
    //then copy matrix to device
    //A* must point to an uninitialized matrix

    FILE* fp;    
    size_t count;
    
    fp = fopen(file,"rb");
    count = fread(A->dim,sizeof(int),2, fp); 
    if(count < 2)
	fprintf(stderr,"read_matrix: fread error\n");

    int N = A->dim[0]*A->dim[1];
    hipHostMalloc((void**)&(A->mat),sizeof(float)*N);
    //A->mat = (float*)malloc(sizeof(float)*A->dim[0]*A->dim[1]);
    count = fread(A->mat,sizeof(float),N,fp);
    if(count < N)
	fprintf(stderr,"read_matrix: fread error\n");
    fclose(fp);

    A->mat_d = NULL;
    copy_matrix_to_device(A);

    printf("read %s [%ix%i]\n",file,A->dim[0],A->dim[1]); 
}

void write_matrix(matrix A, char* file){
    //write matrix to file using column-major order
    //dimensions are written as leading ints

    FILE* fp;    
    size_t count;
    
    fp = fopen(file,"wb");
    count = fwrite(A.dim,sizeof(int),2,fp); 
    if(count < 2)
	fprintf(stderr,"write_matrix: fwrite error\n");

    
    count = fwrite(A.mat,sizeof(float),A.dim[0]*A.dim[1],fp);
    if(count < A.dim[0]*A.dim[1])
	fprintf(stderr,"write_matrix: fwrite error\n");
    fclose(fp);

    printf("write %s [%ix%i]\n",file,A.dim[0],A.dim[1]); 
}

void create_matrix(matrix* A, int rows, int cols, float value){
    //create matrix with all elements equal to 'value'
    //matrix dimensions are in dim (rows,cols)
    //set A->mat_d to NULL
    
    A->dim[0] = rows;
    A->dim[1] = cols;
    const int N = A->dim[0]*A->dim[1];

    A->mat = (float*)malloc(sizeof(float)*N);
    for(int i = 0; i<N; i++)
	A->mat[i] = value;

    if(A->mat_d != NULL)
	hipFree(A->mat_d);

    A->mat_d = NULL;
}

void create_matrix_on_device(matrix* A, int rows, int cols, float value){
    //create matrix on device  with all elements equal to 'value'
    //matrix dimensions are in dim[] {rows,cols}

    A->dim[0] = rows;
    A->dim[1] = cols;
    A->mat = NULL;

    const int N = A->dim[0]*A->dim[1];

    hipError_t err;
    err = hipMalloc((void**) &(A->mat_d), sizeof(float)*N);
    if (err != hipSuccess){
	fprintf(stderr,"create_matrix_on_device: hipMalloc: ErrorMemoryAllocation\n");
	exit(1);
    }

    float *temp = (float*)malloc(sizeof(float)*N);
    for(int i = 0; i<N; i++)
	temp[i] = value;
    hipMemcpy(A->mat_d,temp,sizeof(float)*N,hipMemcpyHostToDevice);

    free(temp);


}

void create_matrix_on_both(matrix* A, int rows, int cols, float value){
    //create matrix on device  with all elements equal to 'value'
    //matrix dimensions are in dim[] {rows,cols}

    A->dim[0] = rows;
    A->dim[1] = cols;
    const int N = A->dim[0]*A->dim[1];
    hipError_t err;


    err = hipMalloc((void**) &(A->mat_d), sizeof(float)*N);
    if (err != hipSuccess){
	fprintf(stderr,"create_matrix_on_both: hipMalloc: ErrorMemoryAllocation\n");
	exit(1);
    }

    A->mat = (float*)malloc(sizeof(float)*N);
    for(int i = 0; i<N; i++)
	A->mat[i] = value;
    hipMemcpy(A->mat_d,A->mat,sizeof(float)*N,hipMemcpyHostToDevice);

}

void destroy_matrix(matrix* A){
    if(A->mat != NULL)
	hipHostFree(A->mat);
    A->mat = NULL;
    if(A->mat_d != NULL)
	hipFree(A->mat_d);
    A->mat_d = NULL;

    A->dim[0] = 0;
    A->dim[1] = 0;
}

void copy_matrix_to_device(matrix* A){

    const int N = A->dim[0]*A->dim[1];
    hipError_t err;

    if (A->mat == NULL){
	fprintf(stderr,"copy_matrix_to_device: matrix not allocated on host\n");
	exit(1);
    }
    if (A->mat_d == NULL){
	err = hipMalloc((void**) &(A->mat_d), sizeof(float)*N);
	if(err != hipSuccess){
	    fprintf(stderr,"copy_matrix_to_device: hipMalloc: FAIL\n");
	    exit(1);
	}
    }

    err = hipMemcpy(A->mat_d,A->mat,sizeof(float)*N, hipMemcpyHostToDevice);
    switch (err){
	case hipErrorInvalidValue:
	fprintf(stderr,"copy_matrix_to_device: hipMemcpy: InvalidValue\n");
	exit(1);
	break;
	case hipErrorInvalidDevicePointer:
	fprintf(stderr,"copy_matrix_to_device: hipMemcpy: InvalidDevicePointer\n");
	exit(1);
	break;
	case hipErrorInvalidMemcpyDirection:
	fprintf(stderr,"copy_matrix_to_device: hipMemcpy: InvalidMemcpyDirection\n");
	exit(1);
	break;
    }
}

void copy_matrix_on_device(matrix A, matrix B){

    if(A.dim[0]!=B.dim[0] || A.dim[1]!=B.dim[1]){
	fprintf(stderr,"copy_matrix_on_device: dimension error\n");
	exit(1);
    }
    const int N = A.dim[0]*A.dim[1];
    hipError_t err;

    if (A.mat_d == NULL){
	fprintf(stderr,"copy_matrix_on_device: source matrix not allocated on device\n");
	exit(1);
    }
    if (B.mat_d == NULL){
	fprintf(stderr,"copy_matrix_on_device: dest. matrix not allocated on device\n");
	exit(1);
    }

    err = hipMemcpy(B.mat_d,A.mat_d,sizeof(float)*N, hipMemcpyDeviceToDevice);
    switch (err){
	case hipErrorInvalidValue:
	fprintf(stderr,"copy_matrix_on_device: hipMemcpy: InvalidValue\n");
	exit(1);
	break;
	case hipErrorInvalidDevicePointer:
	fprintf(stderr,"copy_matrix_on_device: hipMemcpy: InvalidDevicePointer\n");
	exit(1);
	break;
	case hipErrorInvalidMemcpyDirection:
	fprintf(stderr,"copy_matrix_on_device: hipMemcpy: InvalidMemcpyDirection\n");
	exit(1);
	break;
    }
}

void copy_matrix_from_device(matrix* A){

    const int N = A->dim[0]*A->dim[1];

    if (A->mat_d == NULL){
	fprintf(stderr,"copy_matrix_from_device: matrix not allocated on device\n");
	exit(1);
    }
    if (A->mat == NULL)
	hipHostMalloc((void**)&(A->mat),sizeof(float)*N);
	//A->mat = (float*)malloc(sizeof(float)*N);

    hipError_t err;
    err = hipMemcpy(A->mat,A->mat_d,sizeof(float)*N, hipMemcpyDeviceToHost);
    switch (err){
	case hipErrorInvalidValue:
	fprintf(stderr,"copy_matrix_to_device: hipMemcpy: InvalidValue\n");
	exit(1);
	break;
	case hipErrorInvalidDevicePointer:
	fprintf(stderr,"copy_matrix_to_device: hipMemcpy: InvalidDevicePointer\n");
	exit(1);
	break;
	case hipErrorInvalidMemcpyDirection:
	fprintf(stderr,"copy_matrix_to_device: hipMemcpy: InvalidMemcpyDirection\n");
	exit(1);
	break;
    }
}

void print_matrix(matrix A){
    int i,j;
    printf("\n");
    const int lda = A.dim[0];
    const int tda = A.dim[1];
    for(i=0;i<lda;i++){
	for(j=0;j<tda;j++){
	    printf("% 5.5g ",A.mat[i+A.dim[0]*j]);
	}
	printf("\n");
    }
    printf("\n");
}

void matrix_multiply_d( matrix a, matrix b, matrix c ){
    
    hipblasSgemm('N',
	    'N', c.dim[0], c.dim[1],
	    a.dim[1], 1, a.mat_d,
	    a.dim[0], b.mat_d, b.dim[0],
	    0, c.mat_d, c.dim[0]);
    if(cublasGetError() != HIPBLAS_STATUS_SUCCESS){
	fprintf(stderr,"matrix_multiply_d: NOT SUCCESS\n"); 
	exit(1);
    }
}

void matrix_multiply_AtB_d( matrix a, matrix b, matrix c ){
    
    hipblasSgemm('T',
	    'N', c.dim[0], c.dim[1],
	    b.dim[0], 1, a.mat_d,
	    a.dim[0], b.mat_d, b.dim[0],
	    0, c.mat_d, c.dim[0]);
    if(cublasGetError() != HIPBLAS_STATUS_SUCCESS){
	fprintf(stderr,"matrix_multiply_AtB_d: NOT SUCCESS\n"); 
	exit(1);
    }
}

void matrix_multiply_ABt_d( matrix a, matrix b, matrix c ){
    
    hipblasSgemm('N',
	    'T', c.dim[0], c.dim[1],
	    a.dim[1], 1, a.mat_d,
	    a.dim[0], b.mat_d, b.dim[0],
	    0, c.mat_d, c.dim[0]);
    hipblasStatus_t err = cublasGetError();
    if(err != HIPBLAS_STATUS_SUCCESS){
	fprintf(stderr,"matrix_multiply_ABt_d: NOT SUCCESS [%i]\n",err); 
	switch(err){
	    case HIPBLAS_STATUS_NOT_INITIALIZED:
		fprintf(stderr,"HIPBLAS_STATUS_NOT_INITIALIZED\n");
		break;
	    case HIPBLAS_STATUS_ALLOC_FAILED:
		fprintf(stderr,"HIPBLAS_STATUS_ALLOC_FAILED\n");
		break;
	    case HIPBLAS_STATUS_INVALID_VALUE:
		fprintf(stderr,"HIPBLAS_STATUS_INVALID_VALUE\n");
		break;
	    case HIPBLAS_STATUS_MAPPING_ERROR:
		fprintf(stderr,"HIPBLAS_STATUS_MAPPING_ERROR\n");
		break;
	    case HIPBLAS_STATUS_EXECUTION_FAILED:
		fprintf(stderr,"HIPBLAS_STATUS_EXECUTION_FAILED\n");
		break;
	}
	exit(1);
    }
}

float matrix_difference_norm_d(action_t action, matrix a, matrix b, int block1, int lapt1,int block2, int lapt2){
    //memory allocated and not freed
    //block1 - block size for first reduction level
    //block2 - "" for 2nd "" (set to 1 if not using 2nd level)
    //lapt1 - load/adds per thread for first red. lev.
    //lapt2 - "" for 2nd ""
    static int r1size = 0;
    static float *r1 = NULL;
    static float *result_d = NULL;
    if(action==cleanup){
	if(r1!=NULL){
	    hipFree(r1);
	    r1 = NULL;
	}
	if(result_d!=NULL){
	    hipFree(result_d);
	    result_d = NULL;
	}
	r1size = 0;
	return 0;
    }
    
    if(a.dim[0] != b.dim[0] || a.dim[1] != b.dim[1]){
	fprintf(stderr,"matrix_difference_norm_d: dimension error\n");
	exit(1);
    }

    const int N = a.dim[0]*a.dim[1];	//size of each reduction
    
    dim3 dimBlock(block1);
    dim3 dimGrid((N/(block1*lapt1)) + (!(N%(block1*lapt1))?0:1));

    dim3 dimBlock2(block2,1);
    dim3 dimGrid2((dimGrid.x/(block2*lapt2)) + (!(dimGrid.x%(block2*lapt2))?0:1),2);

    //printf("1: %i %i %i %i\n",dimBlock.x,dimBlock.y, dimGrid.x, dimGrid.y);
    //printf("2: %i %i %i %i\n",dimBlock2.x,dimBlock2.y, dimGrid2.x, dimGrid2.y);

    //allocate memory for first level reduction
    if(result_d == NULL)
	hipMalloc((void**) &result_d, sizeof(float)*2);
    if (r1size < dimGrid.x*2){
	if(r1 != NULL)
	    hipFree(r1);
	r1size = dimGrid.x*2;
	hipMalloc((void**) &r1, sizeof(float)*r1size);
    }

    if(block2 <= 1){ //if we only need one level of reduction
	if (dimGrid.x > 1){
	    fprintf(stderr,"matrix_difference_norm_d: dimGrid.x > 1\n");
	    exit(1);
	}
	switch (block1)
	{
	    case 512:
		reduce1DDiff<512><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	    case 256:
		reduce1DDiff<256><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	    case 128:
		reduce1DDiff<128><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	    case 64:
		reduce1DDiff<64><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	    case 32:
		reduce1DDiff<32><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	    case 16:
		reduce1DDiff<16><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	    case 8:
		reduce1DDiff<8><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	}
    }
    else{ //if we need two levels of reduction
	if (dimGrid2.x > 1){
	    fprintf(stderr,"matrix_difference_norm_d: dimGrid2.x > 1\n");
	    exit(1);
	}
	switch (block1)
	{
	    case 512:
		reduce1DDiff<512><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	    case 256:
		reduce1DDiff<256><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	    case 128:
		reduce1DDiff<128><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	    case 64:
		reduce1DDiff<64><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	    case 32:
		reduce1DDiff<32><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	    case 16:
		reduce1DDiff<16><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	    case 8:
		reduce1DDiff<8><<< dimGrid, dimBlock, 2*dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	}
	switch (block2)
	{
	    case 512:
		reduce2D<512><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	    case 256:
		reduce2D<256><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	    case 128:
		reduce2D<128><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	    case 64:
		reduce2D<64><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	    case 32:
		reduce2D<32><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	    case 16:
		reduce2D<16><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	    case 8:
		reduce2D<8><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	}
    }

    float result[2];
    hipMemcpy(result,result_d,2*sizeof(float),hipMemcpyDeviceToHost);
    return result[0]/result[1];



}

float matrix_div_d(action_t action, matrix a, matrix b, int block1, int lapt1,int block2, int lapt2){
    //memory allocated and not freed
    //block1 - block size for first reduction level
    //block2 - "" for 2nd "" (set to 1 if not using 2nd level)
    //lapt1 - load/adds per thread for first red. lev.
    //lapt2 - "" for 2nd ""
    
    static int r1size = 0;
    static float *r1 = NULL;
    static float *result_d = NULL;
    if(action==cleanup){
	if(r1!=NULL){
	    hipFree(r1);
	    r1 = NULL;
	}
	if(result_d!=NULL){
	    hipFree(result_d);
	    result_d = NULL;
	}
	r1size = 0;
	return 0;
    }

    if(a.dim[0] != b.dim[0] || a.dim[1] != b.dim[1]){
	fprintf(stderr,"matrix_div_d: dimension error\n");
	exit(1);
    }

    const int N = a.dim[0]*a.dim[1];	//size of each reduction
    
    dim3 dimBlock(block1);
    dim3 dimGrid((N/(block1*lapt1)) + (!(N%(block1*lapt1))?0:1));

    dim3 dimBlock2(block2);
    dim3 dimGrid2((dimGrid.x/(block2*lapt2)) + (!(dimGrid.x%(block2*lapt2))?0:1));

    //printf("1: %i %i %i %i\n",dimBlock.x,dimBlock.y, dimGrid.x, dimGrid.y);
    //printf("2: %i %i %i %i\n",dimBlock2.x,dimBlock2.y, dimGrid2.x, dimGrid2.y);

    //allocate memory for first level reduction
    if(result_d == NULL)
	hipMalloc((void**) &result_d, sizeof(float)*1);
    if (r1size < dimGrid.x){
	if(r1 != NULL)
	    hipFree(r1);
	r1size = dimGrid.x;
	hipMalloc((void**) &r1, sizeof(float)*r1size);
    }

    if(block2 <= 1){ //if we only need one level of reduction
	if (dimGrid.x > 1){
	    fprintf(stderr,"matrix_difference_norm_d: dimGrid.x > 1\n");
	    exit(1);
	}
	switch (block1)
	{
	    case 512:
		reduce1DDiv<512><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	    case 256:
		reduce1DDiv<256><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	    case 128:
		reduce1DDiv<128><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	    case 64:
		reduce1DDiv<64><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	    case 32:
		reduce1DDiv<32><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	    case 16:
		reduce1DDiv<16><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	    case 8:
		reduce1DDiv<8><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,result_d,N); break;
	}
    }
    else{ //if we need two levels of reduction
	if (dimGrid2.x > 1){
	    fprintf(stderr,"matrix_difference_norm_d: dimGrid2.x > 1\n");
	    exit(1);
	}
	switch (block1)
	{
	    case 512:
		reduce1DDiv<512><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	    case 256:
		reduce1DDiv<256><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	    case 128:
		reduce1DDiv<128><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	    case 64:
		reduce1DDiv<64><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	    case 32:
		reduce1DDiv<32><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	    case 16:
		reduce1DDiv<16><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	    case 8:
		reduce1DDiv<8><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,b.mat_d,r1,N); break;
	}
	switch (block2)
	{
	    case 512:
		reduce2D<512><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	    case 256:
		reduce2D<256><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	    case 128:
		reduce2D<128><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	    case 64:
		reduce2D<64><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	    case 32:
		reduce2D<32><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	    case 16:
		reduce2D<16><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	    case 8:
		reduce2D<8><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,result_d,dimGrid.x); break;
	}
    }

    float result;
    hipMemcpy(&result,result_d,1*sizeof(float),hipMemcpyDeviceToHost);
    return result;



}

void element_divide_d( matrix a, matrix b, matrix c, int block_size){
    // c = a./b

    if(a.dim[0] != b.dim[0] || a.dim[0] != c.dim[0] ||
	    a.dim[1] != b.dim[1] || a.dim[1] != c.dim[1])
    {
	fprintf(stderr,"element_divide_d: dimensions do not agree\n");
	exit(1);
    }

    const int N = a.dim[0]*a.dim[1];
    dim3 dimBlock(block_size);
    dim3 dimGrid((N/dimBlock.x) + (!(N%dimBlock.x)?0:1));
    vecDiv<<<dimGrid,dimBlock>>>(a.mat_d,b.mat_d,c.mat_d,N);
}
    
__global__ void vecDiv(float* a,float* b,float* c,const int N){
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i<N)
	c[i] = __fdividef(a[i],b[i]);
	//c[i] = a[i]/b[i];
}

void element_multiply_d( matrix a, matrix b, matrix c){
    // c = a./b

    if(a.dim[0] == b.dim[0] && a.dim[0] == c.dim[0] &&
	    a.dim[1] == b.dim[1] && a.dim[1] == c.dim[1])
    {
	const int N = a.dim[0]*a.dim[1];
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((N/dimBlock.x) + (!(N%dimBlock.x)?0:1));
	vecMult<<<dimGrid,dimBlock>>>(a.mat_d,b.mat_d,c.mat_d,N);
    }
    else{
	fprintf(stderr,"element_multiply_d: dimensions do not agree\n");
	exit(1);
    }
}
    
__global__ void vecMult(float* a,float* b,float* c,const int N){
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i<N)
	c[i] = a[i]*b[i];
}

__global__ void vecEps(float* a,const int N){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(a[i] < EPS && i < N)
	a[i] = EPS;
}

void matrix_eps_d(matrix a, int block_size){

    const int N = a.dim[0]*a.dim[1];

    dim3 dimBlock(block_size);
    dim3 dimGrid((N/dimBlock.x) + (!(N%dimBlock.x)?0:1));
    vecEps<<<dimGrid, dimBlock>>>(a.mat_d,N);
}

void row_divide_d( matrix a, matrix b, matrix c){
    //element divide every row of 'a' by row vector 'b'

    if(a.dim[1] != b.dim[1] || a.dim[0] != c.dim[0] ||
	    a.dim[1] != c.dim[1] || b.dim[0] != 1){
	fprintf(stderr,"row_divide_d: dimension error\n");
	exit(1);
    }
    int M = a.dim[0]; //number of rows
    int N = a.dim[1]; //number of cols

    dim3 dimBlock(M);
    dim3 dimGrid(N);
    rowDiv<<<dimGrid,dimBlock>>>(a.mat_d,b.mat_d,c.mat_d,M,N);
}

__global__ void rowDiv(float* a, float* b, float* c, int M, int N){

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    c[i] = a[i]/b[blockIdx.x];
}

void col_divide_d( matrix a, matrix b, matrix c){
    //element divide every column of 'a' by column vector 'b'

    if(a.dim[0] != b.dim[0] || a.dim[0] != c.dim[0] ||
	    a.dim[1] != c.dim[1] || b.dim[1] != 1){
	fprintf(stderr,"col_divide: dimension error\n");
	exit(1);
    }
    int M = a.dim[0]; //number of rows
    int N = a.dim[1]; //number of cols
    int block = 32;

    dim3 dimBlock(block,1);
    dim3 dimGrid((M/block) + (!(M%block)?0:1),N);
    colDiv<<<dimGrid,dimBlock>>>(a.mat_d,b.mat_d,c.mat_d,M,N);

}

__global__ void colDiv(float* a, float* b, float* c, int M, int N){

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i<M){
	int ind = i + blockIdx.y*M;
	c[ind] = a[ind]/b[i];
    }
}

__global__ void colMul(float* a, float* b, float* c, int M, int N){

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i<M){
	int ind = i + blockIdx.y*M;
	c[ind] = a[ind]*b[i];
    }
}

void sum_cols_d(action_t action, matrix a, matrix c, int block1, int lapt1,int block2, int lapt2){
    //memory allocated and not freed
    //block1 - block size for first reduction level
    //block2 - "" for 2nd "" (set to 1 if not using 2nd level)
    //lapt1 - load/adds per thread for first red. lev.
    //lapt2 - "" for 2nd ""
    static int r1size = 0;
    static float *r1 = NULL;
    if(action==cleanup){
	if(r1!=NULL){
	    hipFree(r1);
	    r1 = NULL;
	}
	r1size = 0;
	return;
    }
    
    if(a.dim[1] != c.dim[1] || c.dim[0] != 1){
	fprintf(stderr,"sum_cols_d: dimension error\n");
	exit(1);
    }

    const int N = a.dim[0];	//size of each reduction
    const int M = a.dim[1];	//number of reductions
    
    dim3 dimBlock(block1,1);
    dim3 dimGrid((N/(block1*lapt1)) + (!(N%(block1*lapt1))?0:1),M);

    dim3 dimBlock2(block2,1);
    dim3 dimGrid2((dimGrid.x/(block2*lapt2)) + (!(dimGrid.x%(block2*lapt2))?0:1),M);

    //printf("1: %i %i %i %i\n",dimBlock.x,dimBlock.y, dimGrid.x, dimGrid.y);
    //printf("2: %i %i %i %i\n",dimBlock2.x,dimBlock2.y, dimGrid2.x, dimGrid2.y);

    //allocate memory for first level reduction
    if (r1size < dimGrid.x*dimGrid.y){
	if(r1 != NULL)
	    hipFree(r1);
	r1size = dimGrid.x*dimGrid.y;
	hipMalloc((void**) &r1, sizeof(float)*r1size);
    }

    if(block2 <= 1){ //if we only need one level of reduction
	if (dimGrid.x > 1){
	    fprintf(stderr,"sum_cols_d: dimGrid.x > 1\n");
	    exit(1);
	}
	switch (block1)
	{
	    case 512:
		reduce2D<512><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N); break;
	    case 256:
		reduce2D<256><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N); break;
	    case 128:
		reduce2D<128><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N); break;
	    case 64:
		reduce2D<64><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N); break;
	    case 32:
		reduce2D<32><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N); break;
	    case 16:
		reduce2D<16><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N); break;
	    case 8:
		reduce2D<8><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N); break;
	}
    }
    else{ //if we need two levels of reduction
	if (dimGrid2.x > 1){
	    fprintf(stderr,"sum_cols_d: dimGrid2.x > 1\n");
	    exit(1);
	}
	switch (block1)
	{
	    case 512:
		reduce2D<512><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	    case 256:
		reduce2D<256><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	    case 128:
		reduce2D<128><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	    case 64:
		reduce2D<64><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	    case 32:
		reduce2D<32><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	    case 16:
		reduce2D<16><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	    case 8:
		reduce2D<8><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	}
	switch (block2)
	{
	    case 512:
		reduce2D<512><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	    case 256:
		reduce2D<256><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	    case 128:
		reduce2D<128><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	    case 64:
		reduce2D<64><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	    case 32:
		reduce2D<32><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	    case 16:
		reduce2D<16><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	    case 8:
		reduce2D<8><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	}
    }


}

void sum_cols_d_test(action_t action, matrix a, matrix c, int block1, int lapt1,int block2, int lapt2){
    //memory allocated and not freed
    //block1 - block size for first reduction level
    //block2 - "" for 2nd "" (set to 1 if not using 2nd level)
    //lapt1 - load/adds per thread for first red. lev.
    //lapt2 - "" for 2nd ""
    static int r1size = 0;
    static float *r1 = NULL;
    if(action==cleanup){
	if(r1!=NULL){
	    hipFree(r1);
	    r1 = NULL;
	}
	r1size = 0;
	return;
    }
    
    if(a.dim[1] != c.dim[1] || c.dim[0] != 1){
	fprintf(stderr,"sum_cols_d: dimension error\n");
	exit(1);
    }

    const int N = a.dim[0];	//size of each reduction
    const int M = a.dim[1];	//number of reductions
    
    dim3 dimBlock(block1,1);
    dim3 dimGrid((N/(block1*lapt1)) + (!(N%(block1*lapt1))?0:1),1);

    dim3 dimBlock2(block2,1);
    dim3 dimGrid2((dimGrid.x/(block2*lapt2)) + (!(dimGrid.x%(block2*lapt2))?0:1),1);

    //printf("1: %i %i %i %i\n",dimBlock.x,dimBlock.y, dimGrid.x, dimGrid.y);
    //printf("2: %i %i %i %i\n",dimBlock2.x,dimBlock2.y, dimGrid2.x, dimGrid2.y);

    //allocate memory for first level reduction
    if (r1size < dimGrid.x*dimGrid.y){
	if(r1 != NULL)
	    hipFree(r1);
	r1size = dimGrid.x*dimGrid.y;
	hipMalloc((void**) &r1, sizeof(float)*r1size);
    }

    if(block2 <= 1){ //if we only need one level of reduction
	if (dimGrid.x > 1){
	    fprintf(stderr,"sum_cols_d: dimGrid.x > 1\n");
	    exit(1);
	}
	for(int i=0;i<M;i++)
	{
	    switch (block1)
	    {
		case 512:
		    reduce1D<512><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 256:
		    reduce1D<256><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 128:
		    reduce1D<128><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 64:
		    reduce1D<64><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 32:
		    reduce1D<32><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 16:
		    reduce1D<16><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 8:
		    reduce1D<8><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
	    }
	}
    }
    else{ //if we need two levels of reduction
	if (dimGrid2.x > 1){
	    fprintf(stderr,"sum_cols_d: dimGrid2.x > 1\n");
	    exit(1);
	}
	switch (block1)
	{
	    case 512:
		reduce2D<512><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	    case 256:
		reduce2D<256><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	    case 128:
		reduce2D<128><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	    case 64:
		reduce2D<64><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	    case 32:
		reduce2D<32><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	    case 16:
		reduce2D<16><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	    case 8:
		reduce2D<8><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N); break;
	}
	switch (block2)
	{
	    case 512:
		reduce2D<512><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	    case 256:
		reduce2D<256><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	    case 128:
		reduce2D<128><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	    case 64:
		reduce2D<64><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	    case 32:
		reduce2D<32><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	    case 16:
		reduce2D<16><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	    case 8:
		reduce2D<8><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x); break;
	}
    }


}

void sum_cols_d_test2(action_t action, matrix a, matrix c, int block1, int lapt1,int block2, int lapt2){
    //memory allocated and not freed
    //block1 - block size for first reduction level
    //block2 - "" for 2nd "" (set to 1 if not using 2nd level)
    //lapt1 - load/adds per thread for first red. lev.
    //lapt2 - "" for 2nd ""
    static int r1size = 0;
    static float *r1 = NULL;
    if(action==cleanup){
	if(r1!=NULL){
	    hipFree(r1);
	    r1 = NULL;
	}
	r1size = 0;
	return;
    }
    
    if(a.dim[1] != c.dim[1] || c.dim[0] != 1){
	fprintf(stderr,"sum_cols_d: dimension error\n");
	exit(1);
    }

    const int N = a.dim[0];	//size of each reduction
    const int M = a.dim[1];	//number of reductions
    
    dim3 dimBlock(block1,1);
    dim3 dimGrid((N/(block1*lapt1)) + (!(N%(block1*lapt1))?0:1),1);

    dim3 dimBlock2(block2,1);
    dim3 dimGrid2((dimGrid.x/(block2*lapt2)) + (!(dimGrid.x%(block2*lapt2))?0:1),1);

    //printf("1: %i %i %i %i\n",dimBlock.x,dimBlock.y, dimGrid.x, dimGrid.y);
    //printf("2: %i %i %i %i\n",dimBlock2.x,dimBlock2.y, dimGrid2.x, dimGrid2.y);

    //allocate memory for first level reduction
    if (r1size < dimGrid.x*dimGrid.y){
	if(r1 != NULL)
	    hipFree(r1);
	r1size = dimGrid.x*dimGrid.y;
	hipMalloc((void**) &r1, sizeof(float)*r1size);
    }

    if(block2 <= 1){ //if we only need one level of reduction
	if (dimGrid.x > 1){
	    fprintf(stderr,"sum_cols_d: dimGrid.x > 1\n");
	    exit(1);
	}
	for(int i=0;i<M;i++)
	{
	    switch (block1)
	    {
		case 512:
		    reduce1D_rolled<512><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 256:
		    reduce1D_rolled<256><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 128:
		    reduce1D_rolled<128><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 64:
		    reduce1D_rolled<64><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 32:
		    reduce1D_rolled<32><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 16:
		    reduce1D_rolled<16><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 8:
		    reduce1D_rolled<8><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
	    }
	}
    }


}

void sum_cols_d_test3(action_t action, matrix a, matrix c, int block1, int lapt1,int block2, int lapt2){
    //memory allocated and not freed
    //block1 - block size for first reduction level
    //block2 - "" for 2nd "" (set to 1 if not using 2nd level)
    //lapt1 - load/adds per thread for first red. lev.
    //lapt2 - "" for 2nd ""
    static int r1size = 0;
    static float *r1 = NULL;
    if(action==cleanup){
	if(r1!=NULL){
	    hipFree(r1);
	    r1 = NULL;
	}
	r1size = 0;
	return;
    }
    
    if(a.dim[1] != c.dim[1] || c.dim[0] != 1){
	fprintf(stderr,"sum_cols_d: dimension error\n");
	exit(1);
    }

    const int N = a.dim[0];	//size of each reduction
    const int M = a.dim[1];	//number of reductions
    
    dim3 dimBlock(block1,1);
    dim3 dimGrid((N/(block1*lapt1)) + (!(N%(block1*lapt1))?0:1),1);

    dim3 dimBlock2(block2,1);
    dim3 dimGrid2((dimGrid.x/(block2*lapt2)) + (!(dimGrid.x%(block2*lapt2))?0:1),1);

    //printf("1: %i %i %i %i\n",dimBlock.x,dimBlock.y, dimGrid.x, dimGrid.y);
    //printf("2: %i %i %i %i\n",dimBlock2.x,dimBlock2.y, dimGrid2.x, dimGrid2.y);

    //allocate memory for first level reduction
    if (r1size < dimGrid.x*dimGrid.y){
	if(r1 != NULL)
	    hipFree(r1);
	r1size = dimGrid.x*dimGrid.y;
	hipMalloc((void**) &r1, sizeof(float)*r1size);
    }

    if(block2 <= 1){ //if we only need one level of reduction
	if (dimGrid.x > 1){
	    fprintf(stderr,"sum_cols_d: dimGrid.x > 1\n");
	    exit(1);
	}
	for(int i=0;i<M;i++)
	{
	    switch (block1)
	    {
		case 512:
		    reduce1D_rolled2<512><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 256:
		    reduce1D_rolled2<256><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 128:
		    reduce1D_rolled2<128><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 64:
		    reduce1D_rolled2<64><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 32:
		    reduce1D_rolled2<32><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 16:
		    reduce1D_rolled2<16><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 8:
		    reduce1D_rolled2<8><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
	    }
	}
    }


}

void sum_cols_d_test4(action_t action, matrix a, matrix c, int block1, int lapt1,int block2, int lapt2){
    //memory allocated and not freed
    //block1 - block size for first reduction level
    //block2 - "" for 2nd "" (set to 1 if not using 2nd level)
    //lapt1 - load/adds per thread for first red. lev.
    //lapt2 - "" for 2nd ""
    static int r1size = 0;
    static float *r1 = NULL;
    if(action==cleanup){
	if(r1!=NULL){
	    hipFree(r1);
	    r1 = NULL;
	}
	r1size = 0;
	return;
    }
    
    if(a.dim[1] != c.dim[1] || c.dim[0] != 1){
	fprintf(stderr,"sum_cols_d: dimension error\n");
	exit(1);
    }

    const int N = a.dim[0];	//size of each reduction
    const int M = a.dim[1];	//number of reductions
    
    dim3 dimBlock(block1,1);
    dim3 dimGrid((N/(block1*lapt1)) + (!(N%(block1*lapt1))?0:1),1);

    dim3 dimBlock2(block2,1);
    dim3 dimGrid2((dimGrid.x/(block2*lapt2)) + (!(dimGrid.x%(block2*lapt2))?0:1),1);

    //printf("1: %i %i %i %i\n",dimBlock.x,dimBlock.y, dimGrid.x, dimGrid.y);
    //printf("2: %i %i %i %i\n",dimBlock2.x,dimBlock2.y, dimGrid2.x, dimGrid2.y);

    //allocate memory for first level reduction
    if (r1size < dimGrid.x*dimGrid.y){
	if(r1 != NULL)
	    hipFree(r1);
	r1size = dimGrid.x*dimGrid.y;
	hipMalloc((void**) &r1, sizeof(float)*r1size);
    }

    if(block2 <= 1){ //if we only need one level of reduction
	if (dimGrid.x > 1){
	    fprintf(stderr,"sum_cols_d: dimGrid.x > 1\n");
	    exit(1);
	}
	for(int i=0;i<M;i++)
	{
	    switch (block1)
	    {
		case 512:
		    reduce1D_strided<512><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 256:
		    reduce1D_strided<256><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 128:
		    reduce1D_strided<128><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 64:
		    reduce1D_strided<64><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 32:
		    reduce1D_strided<32><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 16:
		    reduce1D_strided<16><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 8:
		    reduce1D_strided<8><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
	    }
	}
    }


}

void sum_cols_d_test5(action_t action, matrix a, matrix c, int block1, int lapt1,int block2, int lapt2){
    //memory allocated and not freed
    //block1 - block size for first reduction level
    //block2 - "" for 2nd "" (set to 1 if not using 2nd level)
    //lapt1 - load/adds per thread for first red. lev.
    //lapt2 - "" for 2nd ""
    static int r1size = 0;
    static float *r1 = NULL;
    if(action==cleanup){
	if(r1!=NULL){
	    hipFree(r1);
	    r1 = NULL;
	}
	r1size = 0;
	return;
    }
    
    if(a.dim[1] != c.dim[1] || c.dim[0] != 1){
	fprintf(stderr,"sum_cols_d: dimension error\n");
	exit(1);
    }

    const int N = a.dim[0];	//size of each reduction
    const int M = a.dim[1];	//number of reductions
    
    dim3 dimBlock(block1,1);
    dim3 dimGrid((N/(block1*lapt1)) + (!(N%(block1*lapt1))?0:1),1);

    dim3 dimBlock2(block2,1);
    dim3 dimGrid2((dimGrid.x/(block2*lapt2)) + (!(dimGrid.x%(block2*lapt2))?0:1),1);

    //printf("1: %i %i %i %i\n",dimBlock.x,dimBlock.y, dimGrid.x, dimGrid.y);
    //printf("2: %i %i %i %i\n",dimBlock2.x,dimBlock2.y, dimGrid2.x, dimGrid2.y);

    //allocate memory for first level reduction
    if (r1size < dimGrid.x*dimGrid.y){
	if(r1 != NULL)
	    hipFree(r1);
	r1size = dimGrid.x*dimGrid.y;
	hipMalloc((void**) &r1, sizeof(float)*r1size);
    }

    if(block2 <= 1){ //if we only need one level of reduction
	if (dimGrid.x > 1){
	    fprintf(stderr,"sum_cols_d: dimGrid.x > 1\n");
	    exit(1);
	}
	for(int i=0;i<M;i++)
	{
	    switch (block1)
	    {
		case 512:
		    reduce1D_interleaved<512><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 256:
		    reduce1D_interleaved<256><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 128:
		    reduce1D_interleaved<128><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 64:
		    reduce1D_interleaved<64><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 32:
		    reduce1D_interleaved<32><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 16:
		    reduce1D_interleaved<16><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
		case 8:
		    reduce1D_interleaved<8><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(&a.mat_d[N*i],&c.mat_d[i],N); break;
	    }
	}
    }


}

void sum_rows_d(action_t action, matrix a, matrix c, int block1, int lapt1, int block2, int lapt2){
    //memory allocated and not freed
    //block1 - block size for first reduction level
    //block2 - "" for 2nd "" (set to 1 if not using 2nd level)
    //lapt1 - load/adds per thread for first red. lev.
    //lapt2 - "" for 2nd ""
    
    static int r1size = 0;
    static float *r1 = NULL;
    if(action==cleanup){
	if(r1!=NULL){
	    hipFree(r1);
	    r1 = NULL;
	}
	r1size = 0;
	return;
    }
    if(a.dim[0] != c.dim[0] || c.dim[1] != 1){
	fprintf(stderr,"sum_rows_d: dimension error\n");
	exit(1);
    }

    const int N = a.dim[1];	//size of each reduction
    const int M = a.dim[0];	//number of reductions
    
    dim3 dimBlock(block1,1);
    dim3 dimGrid((N/(block1*lapt1)) + (!(N%(block1*lapt1))?0:1),M);

    dim3 dimBlock2(block2,1);
    dim3 dimGrid2((dimGrid.x/(block2*lapt2)) + (!(dimGrid.x%(block2*lapt2))?0:1),M);

    //printf("1: %i %i %i %i\n",dimBlock.x,dimBlock.y, dimGrid.x, dimGrid.y);
    //printf("2: %i %i %i %i\n",dimBlock2.x,dimBlock2.y, dimGrid2.x, dimGrid2.y);

    //allocate memory for first level reduction
    if (r1size < dimGrid.x*dimGrid.y){
	if(r1 != NULL)
	    hipFree(r1);
	r1size = dimGrid.x*dimGrid.y;
	hipMalloc((void**) &r1, sizeof(float)*r1size);
    }

    if(block2 <= 1){ //if we only need one level of reduction
	if (dimGrid.x > 1){
	    fprintf(stderr,"sum_rows_d: dimGrid.x > 1\n");
	    exit(1);
	}
	switch (block1)
	{
	    case 512:
		reduce2DStrided<512><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N,M); break;
	    case 256:
		reduce2DStrided<256><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N,M); break;
	    case 128:
		reduce2DStrided<128><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N,M); break;
	    case 64:
		reduce2DStrided<64><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N,M); break;
	    case 32:
		reduce2DStrided<32><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N,M); break;
	    case 16:
		reduce2DStrided<16><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N,M); break;
	    case 8:
		reduce2DStrided<8><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,c.mat_d,N,M); break;
	}
    }
    else{ //if we need two levels of reduction
	if (dimGrid2.x > 1){
	    fprintf(stderr,"sum_rows_d: dimGrid2.x > 1\n");
	    exit(1);
	}
	switch (block1)
	{
	    case 512:
		reduce2DStrided<512><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N,M); break;
	    case 256:
		reduce2DStrided<256><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N,M); break;
	    case 128:
		reduce2DStrided<128><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N,M); break;
	    case 64:
		reduce2DStrided<64><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N,M); break;
	    case 32:
		reduce2DStrided<32><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N,M); break;
	    case 16:
		reduce2DStrided<16><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N,M); break;
	    case 8:
		reduce2DStrided<8><<< dimGrid, dimBlock, dimBlock.x*sizeof(float) >>>(a.mat_d,r1,N,M); break;
	}
	switch (block2)
	{
	    case 512:
		reduce2DStrided<512><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x,M); break;
	    case 256:
		reduce2DStrided<256><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x,M); break;
	    case 128:
		reduce2DStrided<128><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x,M); break;
	    case 64:
		reduce2DStrided<64><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x,M); break;
	    case 32:
		reduce2DStrided<32><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x,M); break;
	    case 16:
		reduce2DStrided<16><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x,M); break;
	    case 8:
		reduce2DStrided<8><<< dimGrid2, dimBlock2, dimBlock2.x*sizeof(float) >>>(r1,c.mat_d,dimGrid.x,M); break;
	}
    }


}


template <unsigned int blockSize>
__global__ void reduce1DDiff(float *g_idata1, float *g_idata2, float *g_odata, int N){
    extern __shared__ float sdata[];
    float* diff = (float*)sdata;
    float* sum = (float*)&sdata[blockSize];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    int i = blockIdx.x*blockSize + threadIdx.x;
    const int gridSize = blockSize*gridDim.x;
    sum[tid] = 0;
    diff[tid] = 0;
    while (i < N) { 
	diff[tid] += fabs(g_idata1[i] - g_idata2[i]);
	sum[tid] += fabs(g_idata1[i]);
       	i += gridSize; 
    }
    __syncthreads();
    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { diff[tid] += diff[tid + 256]; sum[tid] += sum[tid + 256]; }
       	__syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { diff[tid] += diff[tid + 128]; sum[tid] += sum[tid + 128]; } 
	__syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { diff[tid] += diff[tid + 64]; sum[tid] += sum[tid + 64]; } 
	__syncthreads(); }
    if (tid < 32) {
	if (blockSize >= 64){ diff[tid] += diff[tid + 32]; sum[tid] += sum[tid + 32]; }
	if (blockSize >= 32){ diff[tid] += diff[tid + 16]; sum[tid] += sum[tid + 16]; }
	if (blockSize >= 16){ diff[tid] += diff[tid + 8]; sum[tid] += sum[tid + 8]; }
	if (blockSize >= 8){ diff[tid] += diff[tid + 4]; sum[tid] += sum[tid + 4]; }
	if (blockSize >= 4){ diff[tid] += diff[tid + 2]; sum[tid] += sum[tid + 2]; }
	if (blockSize >= 2){ diff[tid] += diff[tid + 1]; sum[tid] += sum[tid + 1]; }
    }

    // write result for this block to global mem
    if (tid == 0){ 
	g_odata[blockIdx.x + gridDim.x] = sum[0];
	g_odata[blockIdx.x] = diff[0];
    }
}

template <unsigned int blockSize>
__global__ void reduce1DDiv(float *g_idata1, float *g_idata2, float *g_odata, int N){
    extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    int i = blockIdx.x*blockSize + threadIdx.x;
    const int gridSize = blockSize*gridDim.x;
    float x;
    float y;
    sdata[tid] = 0;
    while (i < N) { 
	x = g_idata1[i];
	y = g_idata2[i];
	//sdata[tid] += (x*__logf(x/y)-x+y); 
	sdata[tid] += (x*(__logf(x)-__logf(y))-x+y); 
       	i += gridSize; 
    }
    __syncthreads();
    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; }
       	__syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; }
	__syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; }
	__syncthreads(); }
    if (tid < 32) {
	if (blockSize >= 64){ sdata[tid] += sdata[tid + 32]; }
	if (blockSize >= 32){ sdata[tid] += sdata[tid + 16]; }
	if (blockSize >= 16){ sdata[tid] += sdata[tid + 8]; }
	if (blockSize >= 8){ sdata[tid] += sdata[tid + 4]; }
	if (blockSize >= 4){ sdata[tid] += sdata[tid + 2]; }
	if (blockSize >= 2){ sdata[tid] += sdata[tid + 1]; }
    }

    // write result for this block to global mem
    if (tid == 0){ 
	g_odata[blockIdx.x] = sdata[0];
    }
}

template <unsigned int blockSize>
__global__ void reduce2D(float *g_idata, float *g_odata, int N){
    extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    int i = blockIdx.x*blockSize*2 + threadIdx.x;
    const unsigned int offset = blockIdx.y*N;
    const unsigned int gridSize = blockSize*2*gridDim.x;
    int n = N - blockSize;
    sdata[tid] = 0;
    while (i < n) { sdata[tid] += g_idata[i+offset] + g_idata[i+offset+blockSize]; i += gridSize; }
    if(i<N)
	sdata[tid] += g_idata[i+offset];
    __syncthreads();
    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) {
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x + blockIdx.y*gridDim.x] = sdata[0];
}

template <unsigned int blockSize>
__global__ void reduce1D(float *g_idata, float *g_odata, int N){
    extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    int i = blockIdx.x*blockSize*2 + threadIdx.x;
    const unsigned int gridSize = blockSize*2*gridDim.x;
    int n = N - blockSize;
    sdata[tid] = 0;
    while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+blockSize]; i += gridSize; }
    if(i<N)
	sdata[tid] += g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) {
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize>
__global__ void reduce1D_rolled(float *g_idata, float *g_odata, int N){
    extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    int i = blockIdx.x*blockSize*2 + threadIdx.x;
    const unsigned int gridSize = blockSize*2*gridDim.x;
    int n = N - blockSize;
    sdata[tid] = 0;
    while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+blockSize]; i += gridSize; }
    if(i<N)
	sdata[tid] += g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>32; s>>=1)
    {
	if (tid < s)
	    sdata[tid] += sdata[tid + s];
	__syncthreads();
    }
    if (tid < 32)
    {
	sdata[tid] += sdata[tid + 32];
	sdata[tid] += sdata[tid + 16];
	sdata[tid] += sdata[tid + 8];
	sdata[tid] += sdata[tid + 4];
	sdata[tid] += sdata[tid + 2];
	sdata[tid] += sdata[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize>
__global__ void reduce1D_rolled2(float *g_idata, float *g_odata, int N){
    extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    int i = blockIdx.x*blockSize*2 + threadIdx.x;
    const unsigned int gridSize = blockSize*2*gridDim.x;
    int n = N - blockSize;
    sdata[tid] = 0;
    while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+blockSize]; i += gridSize; }
    if(i<N)
	sdata[tid] += g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
	if (tid < s)
	    sdata[tid] += sdata[tid + s];
	__syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize>
__global__ void reduce1D_strided(float *g_idata, float *g_odata, int N){
    extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    int i = blockIdx.x*blockSize*2 + threadIdx.x;
    const unsigned int gridSize = blockSize*2*gridDim.x;
    int n = N - blockSize;
    sdata[tid] = 0;
    while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+blockSize]; i += gridSize; }
    if(i<N)
	sdata[tid] += g_idata[i];
    __syncthreads();
    // do reduction in shared mem

    for (unsigned int s=1; s < blockDim.x; s *= 2) {
	int index = 2 * s * tid;
	if (index < blockDim.x) {
	    sdata[index] += sdata[index + s];
	}
	__syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize>
__global__ void reduce1D_interleaved(float *g_idata, float *g_odata, int N){
    extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    int i = blockIdx.x*blockSize*2 + threadIdx.x;
    const unsigned int gridSize = blockSize*2*gridDim.x;
    int n = N - blockSize;
    sdata[tid] = 0;
    while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+blockSize]; i += gridSize; }
    if(i<N)
	sdata[tid] += g_idata[i];
    __syncthreads();
    // do reduction in shared mem

    for (unsigned int s=1; s < blockDim.x; s *= 2) {
	if (tid % (2*s) == 0) {
	    sdata[tid] += sdata[tid + s];
	}
	__syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize>
__global__ void reduce2DStrided(float *g_idata, float *g_odata, int N, int stride){
    extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    int i = blockIdx.x*blockSize*2 + threadIdx.x;
    const unsigned int offset = blockIdx.y;
    const unsigned int gridSize = blockSize*2*gridDim.x;
    int n = N - blockSize;
    sdata[tid] = 0;
    while (i < n) { 
	sdata[tid] += g_idata[i*stride+offset] + g_idata[(i+blockSize)*stride+offset];
       	i += gridSize; 
    }
    if(i<N)
	sdata[tid] += g_idata[i*stride+offset];
    __syncthreads();
    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) {
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.y + blockIdx.x*gridDim.y] = sdata[0];
}



